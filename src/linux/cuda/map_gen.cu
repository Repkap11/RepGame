#include "hip/hip_runtime.h"
#include <stdlib.h>

// #include "common/RepGame.hpp"
#include "common/map_gen.hpp"
#include "common/block_definitions.hpp"
#include "linux/cuda/perlin_noise.hpp"
#include "common/chunk_loader.hpp"
#include "common/chunk.hpp"

__device__ float map_gen_hills_cuda( int x, int z ) {
    float noise = perlin_noise2d_cuda( x, z, 0.02f, 3, MAP_SEED );
    return ( noise - 0.5f ) * 15;
}

__device__ float map_gen_ground_noise_cuda( int x, int z ) {
    float noise = perlin_noise2d_cuda( x, z, 0.1f, 2, MAP_SEED + 1 );
    return ( noise - 0.5f ) * 2;
}

__device__ float map_gen_mountians_cuda( int x, int z ) {
    float noise = perlin_noise2d_cuda( x, z, 0.008f, 3, MAP_SEED + 2 );
    noise = noise - 0.5f;
    if ( noise < 0 ) {
        noise = 0;
    }
    float mountians = noise * noise * noise * 1000;
    return mountians;
}

__device__ float map_gen_mountian_block_cuda( int x, int z ) {
    float noise = perlin_noise2d_cuda( x, z, 0.4f, 2, MAP_SEED + 3 );
    return noise;
}

__device__ float map_gen_under_water_block_cuda( int x, int z ) {
    float noise = perlin_noise2d_cuda( x, z, 0.2f, 2, MAP_SEED + 4 );
    return noise;
}

__device__ float map_gen_level_cuda( int x, int z ) {
    float noise_orig = perlin_noise2d_cuda( x, z, 0.004f, 2, MAP_SEED + 5 );
    noise_orig = ( noise_orig - 0.5f ) * 10;
    float noise = fabs( noise_orig );
    noise = noise * noise_orig;
    noise = noise > 1 ? 1 : noise;
    noise = noise < -1 ? -1 : noise;

    return noise * 10;
}

__device__ float map_gen_cave_density_cuda( int x, int y, int z ) {
    float noise = perlin_noise3d_cuda( x, y, z, 0.03f, 3, MAP_SEED + 6 );
    return noise;
}

__device__ float map_gen_inverse_lerp_cuda( float min, float max, float value ) {
    if ( value < min ) {
        return 0.0f;
    }
    if ( value > max ) {
        return 1.0f;
    }
    return ( value - min ) / ( max - min );
}


#define MAP_GEN( func, ... ) map_gen_##func##_cuda( __VA_ARGS__ )

__global__ void cuda_set_block( BlockState *blocks, int chunk_x, int chunk_y, int chunk_z ) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if ( index < CHUNK_BLOCK_SIZE ) {
        int y = ( index / ( CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL ) ) - 1;
        int x = ( ( index / CHUNK_SIZE_INTERNAL ) % CHUNK_SIZE_INTERNAL ) - 1;
        int z = ( index % ( CHUNK_SIZE_INTERNAL ) ) - 1;
        x += chunk_x;
        y += chunk_y;
        z += chunk_z;

        float ground_noise = map_gen_ground_noise_cuda( x, z );
        float hills = map_gen_hills_cuda( x, z );
        float mountians = map_gen_mountians_cuda( x, z );
        float level = map_gen_level_cuda( x, z );
        float terrainHeight = level + mountians + hills + ground_noise;
#include "common/map_logic.hpp"

        blocks[ index ] = {finalBlockId, BLOCK_ROTATE_0, 0, finalBlockId};//Assumes all blocks don't spawn with redstone power
    }
}

#define NUM_THREADS_PER_BLOCK 256

__host__ void map_gen_load_block_cuda( Chunk *chunk ) {

    BlockState *device_blocks;
    hipMalloc( &device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockState ) );

    cuda_set_block<<<( CHUNK_BLOCK_SIZE + ( NUM_THREADS_PER_BLOCK - 1 ) ) / NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK, 0>>>( device_blocks, chunk->chunk_x * CHUNK_SIZE, chunk->chunk_y * CHUNK_SIZE, chunk->chunk_z * CHUNK_SIZE );

    hipMemcpy( chunk->blocks, device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockState ), hipMemcpyDeviceToHost );
    hipFree( device_blocks );
}