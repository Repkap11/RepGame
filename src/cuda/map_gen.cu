#include "hip/hip_runtime.h"
#include "RepGame.h"
#include "map_gen.h"
#include "block_definitions.h"
#include "cuda/perlin_noise.h"

__device__ float map_gen_hills_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.02f, 3, MAP_SEED );
    return ( noise - 0.5f ) * 15;
}

__global__ void cuda_set_block(BlockID* blocks, int chunk_x, int chunk_y, int chunk_z){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int y = ( index / ( CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL ) ) - 1;
    int x = ( ( index / CHUNK_SIZE_INTERNAL ) % CHUNK_SIZE_INTERNAL ) - 1;
    int z = ( index % ( CHUNK_SIZE_INTERNAL ) ) - 1;

    x += chunk_x;
    y += chunk_y;
    z += chunk_z;

    float height = map_gen_hills_cuda(x, z);

    BlockID finalBlockId = AIR;
    if (y < height){
        finalBlockId = GRASS;
    }
    blocks[index] = finalBlockId;
}

#define NUM_THREADS_PER_BLOCK 256

__host__ void map_gen_load_block_cuda( Chunk *chunk ) {

    BlockID* device_blocks;
    hipMalloc(&device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID ));

    cuda_set_block<<<(CHUNK_BLOCK_SIZE + (NUM_THREADS_PER_BLOCK-1))/NUM_THREADS_PER_BLOCK , NUM_THREADS_PER_BLOCK>>>(device_blocks,
        chunk->chunk_x * CHUNK_SIZE,
        chunk->chunk_y * CHUNK_SIZE,
        chunk->chunk_z * CHUNK_SIZE);

    hipMemcpy(chunk->blocks, device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID) , hipMemcpyDeviceToHost);
    hipFree(device_blocks);
}