#include "hip/hip_runtime.h"
#include "RepGame.h"
#include "map_gen.h"
#include "block_definitions.h"
#include "cuda/perlin_noise.h"
#include "chunk_loader.h"
#include "chunk.h"

#include <stdlib.h>
#define WATER_LEVEL 0
#define MOUNTAN_CAP_HEIGHT 50

__device__ float map_gen_hills( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.02f, 3, MAP_SEED );
    return ( noise - 0.5f ) * 15;
}

__device__ float map_gen_ground_noise( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.1f, 2, MAP_SEED + 1 );
    return ( noise - 0.5f ) * 2;
}

__device__ float map_gen_level( int x, int z ) {
    float noise_orig = perlin_noise_cuda( x, z, 0.004f, 2, MAP_SEED + 5 );
    noise_orig = ( noise_orig - 0.5f ) * 10;
    float noise = fabs( noise_orig );
    noise = noise * noise_orig;
    noise = noise > 1 ? 1 : noise;
    noise = noise < -1 ? -1 : noise;

    return noise * 10;
}

__device__ float map_gen_mountians( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.008f, 3, MAP_SEED + 2 );
    noise = noise - 0.5f;
    if ( noise < 0 ) {
        noise = 0;
    }
    float mountians = noise * noise * noise * 1000;
    return mountians;
}

__device__ float map_gen_mountian_block( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.8f, 8, MAP_SEED + 3 );
    return noise;
}

__device__ float map_gen_under_water_block( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.2f, 2, MAP_SEED + 4 );
    return noise;
}

__global__ void cuda_set_block(BlockID* blocks, int chunk_x, int chunk_y, int chunk_z){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < CHUNK_BLOCK_SIZE){
        int y = ( index / ( CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL ) ) - 1;
        int x = ( ( index / CHUNK_SIZE_INTERNAL ) % CHUNK_SIZE_INTERNAL ) - 1;
        int z = ( index % ( CHUNK_SIZE_INTERNAL ) ) - 1;
        x += chunk_x;
        y += chunk_y;
        z += chunk_z;

        float ground_noise = map_gen_ground_noise(x, z);
        float hills = map_gen_hills(x, z);
        float mountians = map_gen_mountians( x, z);
        float level = map_gen_level(x, z);
        float terrainHeight = level + mountians + hills + ground_noise;
#include "map_logic.h"

        blocks[index] = finalBlockId;
    }
}

#define NUM_THREADS_PER_BLOCK 256

__host__ void map_gen_load_block_cuda( Chunk *chunk ) {

    BlockID* device_blocks;
    hipMalloc(&device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID ));

    cuda_set_block<<<(CHUNK_BLOCK_SIZE + (NUM_THREADS_PER_BLOCK-1))/NUM_THREADS_PER_BLOCK , NUM_THREADS_PER_BLOCK, 0>>>(device_blocks,
        chunk->chunk_x * CHUNK_SIZE,
        chunk->chunk_y * CHUNK_SIZE,
        chunk->chunk_z * CHUNK_SIZE);

    hipMemcpy(chunk->blocks, device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID) , hipMemcpyDeviceToHost);
    hipFree(device_blocks);
}