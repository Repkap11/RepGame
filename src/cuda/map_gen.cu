#include "hip/hip_runtime.h"
#include "RepGame.h"
#include "map_gen.h"
#include "block_definitions.h"

__global__ void cuda_set_block(BlockID* blocks, int chunk_x, int chunk_y, int chunk_z){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int y = ( index / ( CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL ) ) - 1;
    int x = ( ( index / CHUNK_SIZE_INTERNAL ) % CHUNK_SIZE_INTERNAL ) - 1;
    int z = ( index % ( CHUNK_SIZE_INTERNAL ) ) - 1;

    //x += chunk_x;
    y += chunk_y;
    //z += chunk_z;

    BlockID finalBlockId = AIR;
    if (y < x+z){
        finalBlockId = GRASS;
    }
    blocks[index] = finalBlockId;
}

#define NUM_THREADS_PER_BLOCK 256

__host__ void map_gen_load_block_cuda( Chunk *chunk ) {

    BlockID* device_blocks;
    hipMalloc(&device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID ));

    cuda_set_block<<<(CHUNK_BLOCK_SIZE + (NUM_THREADS_PER_BLOCK-1))/NUM_THREADS_PER_BLOCK , NUM_THREADS_PER_BLOCK>>>(device_blocks,
        chunk->chunk_x * CHUNK_SIZE,
        chunk->chunk_y * CHUNK_SIZE,
        chunk->chunk_z * CHUNK_SIZE);

    hipMemcpy(chunk->blocks, device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID) , hipMemcpyDeviceToHost);
    hipFree(device_blocks);
}