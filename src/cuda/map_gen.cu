#include "hip/hip_runtime.h"
#include "RepGame.h"
#include "map_gen.h"
#include "block_definitions.h"
#include "cuda/perlin_noise.h"
#include "chunk_loader.h"
#include "chunk.h"

#include <stdlib.h>
#define WATER_LEVEL 0
#define MOUNTAN_CAP_HEIGHT 50

__device__ float map_gen_hills_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.02f, 3, MAP_SEED );
    return ( noise - 0.5f ) * 15;
}

__device__ float map_gen_ground_noise_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.1f, 2, MAP_SEED + 1 );
    return ( noise - 0.5f ) * 2;
}

__device__ float map_gen_level_cuda( int x, int z ) {
    float noise_orig = perlin_noise_cuda( x, z, 0.004f, 2, MAP_SEED + 5 );
    noise_orig = ( noise_orig - 0.5f ) * 10;
    float noise = fabs( noise_orig );
    noise = noise * noise_orig;
    noise = noise > 1 ? 1 : noise;
    noise = noise < -1 ? -1 : noise;

    return noise * 10;
}

__device__ float map_gen_mountians_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.008f, 3, MAP_SEED + 2 );
    noise = noise - 0.5f;
    if ( noise < 0 ) {
        noise = 0;
    }
    float mountians = noise * noise * noise * 1000;
    return mountians;
}

__device__ float map_gen_mountian_block_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.8f, 8, MAP_SEED + 3 );
    return noise;
}

__device__ float map_gen_under_water_block_cuda( int x, int z ) {
    float noise = perlin_noise_cuda( x, z, 0.2f, 2, MAP_SEED + 4 );
    return noise;
}

__device__ int chunk_get_index_from_coords_cuda( int x, int y, int z ) {
    return ( y + 1 ) * CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL + ( x + 1 ) * CHUNK_SIZE_INTERNAL + ( z + 1 );
}

#define MAP_GEN(func, ...) map_gen_##func##_cuda(__VA_ARGS__)

__global__ void cuda_set_block(BlockID* blocks, int chunk_x, int chunk_y, int chunk_z){
    int val = blockIdx.x * blockDim.x + threadIdx.x;
    if (val < CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL){
        int block_x = val / CHUNK_SIZE_INTERNAL - 1;
        int block_z = val % CHUNK_SIZE_INTERNAL - 1;
        int x = block_x + chunk_x;
        int z = block_z + chunk_z;

        float ground_noise = map_gen_ground_noise_cuda(x, z);
        float hills = map_gen_hills_cuda(x, z);
        float mountians = map_gen_mountians_cuda( x, z);
        float level = map_gen_level_cuda(x, z);
        float terrainHeight = level + mountians + hills + ground_noise;
        for ( int block_y = -1; block_y < CHUNK_SIZE_INTERNAL - 1; block_y++ ) {
            int y = block_y + chunk_y;
            int index = chunk_get_index_from_coords_cuda( block_x, block_y, block_z );
        #include "map_logic.h"
            blocks[index] = finalBlockId;
        }
    }
}

#define NUM_THREADS_PER_BLOCK 256

__host__ void map_gen_load_block_cuda( Chunk *chunk ) {

    BlockID* device_blocks;
    hipMalloc(&device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID ));

    cuda_set_block<<<(CHUNK_SIZE_INTERNAL * CHUNK_SIZE_INTERNAL + (NUM_THREADS_PER_BLOCK-1))/NUM_THREADS_PER_BLOCK , NUM_THREADS_PER_BLOCK, 0>>>(device_blocks,
        chunk->chunk_x * CHUNK_SIZE,
        chunk->chunk_y * CHUNK_SIZE,
        chunk->chunk_z * CHUNK_SIZE);

    hipMemcpy(chunk->blocks, device_blocks, CHUNK_BLOCK_SIZE * sizeof( BlockID) , hipMemcpyDeviceToHost);
    hipFree(device_blocks);
}