#include "hip/hip_runtime.h"
#include "RepGame.hpp"
#include "map_gen.hpp"
#include "block_definitions.hpp"

__device__ static int hash[] = {208, 34,  231, 213, 32,  248, 233, 56,  161, 78,  24,  140, 71,  48,  140, 254, 245, 255, 247, 247, 40,  185, 248, 251, 245, 28,  124, 204, 204, 76,  36,  1,   107, 28,  234, 163, 202, 224, 245, 128, 167, 204, 9,
    92,  217, 54,  239, 174, 173, 102, 193, 189, 190, 121, 100, 108, 167, 44,  43,  77,  180, 204, 8,   81,  70,  223, 11,  38,  24,  254, 210, 210, 177, 32,  81,  195, 243, 125, 8,   169, 112, 32,  97,  53,  195, 13,
    203, 9,   47,  104, 125, 117, 114, 124, 165, 203, 181, 235, 193, 206, 70,  180, 174, 0,   167, 181, 41,  164, 30,  116, 127, 198, 245, 146, 87,  224, 149, 206, 57,  4,   192, 210, 65,  210, 129, 240, 178, 105, 228,
    108, 245, 148, 140, 40,  35,  195, 38,  58,  65,  207, 215, 253, 65,  85,  208, 76,  62,  3,   237, 55,  89,  232, 50,  217, 64,  244, 157, 199, 121, 252, 90,  17,  212, 203, 149, 152, 140, 187, 234, 177, 73,  174,
    193, 100, 192, 143, 97,  53,  145, 135, 19,  103, 13,  90,  135, 151, 199, 91,  239, 247, 33,  39,  145, 101, 120, 99,  3,   186, 86,  99,  41,  237, 203, 111, 79,  220, 135, 158, 42,  30,  154, 120, 67,  87,  167,
    135, 176, 183, 191, 253, 115, 184, 21,  233, 58,  129, 233, 142, 39,  128, 211, 118, 137, 139, 255, 114, 20,  218, 113, 154, 27,  127, 246, 250, 1,   8,   198, 250, 209, 92,  222, 173, 21,  88,  102, 219};

__device__ int noise2_cuda( int x, int y, int seed ) {
    int tmp = hash[ ( unsigned int )( y + seed ) % 256 ];
    return hash[ ( unsigned int )( tmp + x ) % 256 ];
}

__device__ float lin_inter_cuda( float x, float y, float s ) {
    return x + s * ( y - x );
}

__device__ float smooth_inter_cuda( float x, float y, float s ) {
    return lin_inter_cuda( x, y, s * s * ( 3 - 2 * s ) );
}

__device__ float noise2d_cuda( float x, float y, int seed ) {
    int x_int = floor( x );
    int y_int = floor( y );
    float x_frac = x - x_int;
    float y_frac = y - y_int;
    int s = noise2_cuda( x_int, y_int, seed );
    int t = noise2_cuda( x_int + 1, y_int, seed );
    int u = noise2_cuda( x_int, y_int + 1, seed );
    int v = noise2_cuda( x_int + 1, y_int + 1, seed );
    float low = smooth_inter_cuda( s, t, x_frac );
    float high = smooth_inter_cuda( u, v, x_frac );
    return smooth_inter_cuda( low, high, y_frac );
}

__device__ float perlin_noise_cuda( float x, float y, float freq, int depth, int seed ){
    float xa = x * freq;
    float ya = y * freq;
    float amp = 1.0;
    float fin = 0;
    float div = 0.0;
    int i;
    for ( i = 0; i < depth; i++ ) {
        div += 256 * amp;
        fin += noise2d_cuda( xa, ya, seed ) * amp;
        amp /= 2;
        xa *= 2;
        ya *= 2;
    }
    return fin / div;
}